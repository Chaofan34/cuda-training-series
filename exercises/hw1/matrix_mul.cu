#include "hip/hip_runtime.h"
#include <stdio.h>

// these are just for timing measurments
#include <time.h>
#include "include/error.h"
#include "include/timer.h"

const int DSIZE = 4096;
const int block_size = 16; // CUDA maximum is 1024 *total* threads in block
const float A_val = 1.0f;
const float B_val = 2.0f;

// matrix multiply (naive) kernel: C = A * B
__global__ void mmul(const float *A, const float *B, float *C, int ds)
{

  int idx = threadIdx.x + blockDim.x * blockIdx.x; // create thread x index
  int idy = threadIdx.y + blockDim.y * blockIdx.y; // create thread y index

  if ((idx < ds) && (idy < ds))
  {
    float temp = 0;
    for (int i = 0; i < ds; i++)
      temp += A[idx * ds + i] * B[i * ds + idy]; // dot product of row and column
    C[idy * ds + idx] = temp;
  }
}

void mmul_host(const float *A, const float *B, float *C, int ds)
{
  for (int i = 0; i < ds; i++)
  {
    // printf("mul_host: i:%d\n", i);
    for (int j = 0; j < ds; j++)
      for (int k = 0; k < ds; k++)
        C[i * ds + j] += A[i * ds + k] * B[k * ds + j];
  }
}

int main()
{

  float *h_A, *h_B, *h_C, *h_D, *d_A, *d_B, *d_C;

  // these are just for timing

  {
    auto timer = TimeMonitor("Begin Compute");
    h_A = new float[DSIZE * DSIZE];
    h_B = new float[DSIZE * DSIZE];
    h_C = new float[DSIZE * DSIZE];
    h_D = new float[DSIZE * DSIZE];
    for (int i = 0; i < DSIZE * DSIZE; i++)
    {
      h_A[i] = A_val;
      h_B[i] = B_val;
      h_C[i] = 0;
    }
  }

  {
    auto timer = TimeMonitor("GPU Compute");
    // Allocate device memory and copy input data over to GPU
    hipMalloc(&d_A, DSIZE * DSIZE * sizeof(float));
    hipMalloc(&d_B, DSIZE * DSIZE * sizeof(float));
    hipMalloc(&d_C, DSIZE * DSIZE * sizeof(float));
    cudaCheckErrors("hipMalloc failure");
    hipMemcpy(d_A, h_A, DSIZE * DSIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE * DSIZE * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");

    // Cuda processing sequence step 1 is complete

    // Launch kernel
    dim3 block(block_size, block_size); // dim3 variable holds 3 dimensions
    dim3 grid((DSIZE + block.x - 1) / block.x, (DSIZE + block.y - 1) / block.y);
    mmul<<<grid, block>>>(d_A, d_B, d_C, DSIZE);
    cudaCheckErrors("kernel launch failure");

    // Cuda processing sequence step 2 is complete
    // Copy results back to host
    hipMemcpy(h_C, d_C, DSIZE * DSIZE * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
  }

  // CPU Compute
  {
    auto timer = TimeMonitor("CPU Compute");
    mmul_host(h_A, h_B, h_D, DSIZE);
  }

  // Verify results
  for (int i = 0; i < DSIZE * DSIZE; i++)
  {
    if (h_C[i] != A_val * B_val * DSIZE)
    {
      printf("mismatch at index %d, was: %f, should be: %f\n", i, h_C[i], A_val * B_val * DSIZE);
      return -1;
    }
    if (h_D[i] != h_C[i])
    {
      printf("mismatch at index %d, was: %f, should be: %f\n", i, h_D[i], h_C[i]);
      return -1;
    }
  }
  printf("Success!\n");
  return 0;
}

// $ ./matrix_mul, 没有开优化，CPU频率3GHZ，不懂为啥需要978s
// TimeMonitor::Begin Compute, took 0.056859 seconds
// TimeMonitor::GPU Compute, took 0.783124 seconds
// TimeMonitor::CPU Compute, took 978.699 seconds